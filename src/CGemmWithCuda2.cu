#include <hip/hip_runtime.h>
// #include <stdio.h>

__global__ void gemm_kernel(int M, int N, int K, float alpha, float *A,
                            float *B, float beta, float *C) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < M && col < N) {
    float temp = 0.0;
    for (int k = 0; k < K; k++) {
      temp += A[row * K + k] * B[k * N + col];
    }
    C[row * N + col] = alpha * temp + beta * C[row * N + col];
  }
}

void gemm_cuda(int M, int N, int K, float alpha, float *A, float *B, float beta,
               float *C) {
  float *d_A, *d_B, *d_C;

  // 分配 GPU 内存
  hipMalloc((void **)&d_A, M * K * sizeof(float));
  hipMalloc((void **)&d_B, K * N * sizeof(float));
  hipMalloc((void **)&d_C, M * N * sizeof(float));

  // 复制矩阵到 GPU
  hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, K * N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_C, C, M * N * sizeof(float), hipMemcpyHostToDevice);

  // 定义线程块和网格
  dim3 blockSize(16, 16);
  dim3 gridSize((N + blockSize.x - 1) / blockSize.x,
                (M + blockSize.y - 1) / blockSize.y);

  // 调用 CUDA kernel
  gemm_kernel<<<gridSize, blockSize>>>(M, N, K, alpha, d_A, d_B, beta, d_C);

  // 复制结果回 CPU
  hipMemcpy(C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

  // 释放 GPU 内存
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
