#include "hip/hip_runtime.h"
#include "../include/CGemmWithCuda.h"
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

class Vector {
private:
    float* data;
    int size;

public:
    // 构造函数（只在主机上调用）
    __host__ Vector(int size) : size(size) {
        hipMallocManaged(&data, size * sizeof(float));
    }

    // 析构函数（只在主机上调用）
    __host__ ~Vector() {
        hipFree(data);
    }

    // 访问元素（设备和主机都可调用）
    __host__ __device__ float& operator[](int index) {
        return data[index];
    }

    // 获取大小（设备和主机都可调用）
    __host__ __device__ int getSize() const {
        return size;
    }

    // 设备端向量加法核函数
    __device__ void add(Vector& other) {
        int idx = threadIdx.x + blockIdx.x * blockDim.x;
        if (idx < size) {
            data[idx] += other[idx];
        }
    }

    // 启动内核的主机方法
    __host__ void launchAddKernel(Vector& other) {
        int threadsPerBlock = 256;
        int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
        addKernel<<<blocksPerGrid, threadsPerBlock>>>(*this, other);
        hipDeviceSynchronize();
    }

    // 内核函数
    static __global__ void addKernel(Vector a, Vector b) {
        a.add(b);
    }

};